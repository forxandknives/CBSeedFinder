#include "hip/hip_runtime.h"
﻿#include "Random.cuh"
#include "CreateSeed.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "float.h"

#include <stdio.h>

#include <iostream>

__device__ void dummy();

__global__ void testFunction(int* outputArray);

int main()
{
    constexpr uint64_t TOTAL_SEEDS = 512;//(1ULL << 31);
    constexpr uint32_t THREADS_PER_BLOCK = 256;
    constexpr uint32_t BLOCKS_PER_RUN = TOTAL_SEEDS / THREADS_PER_BLOCK;
  
    hipError_t c;

    //_control87();

    const int arraySize = 1;    
    int* cudaOutput = 0;
    int* output = (int*)malloc(sizeof(int) * arraySize);   

    c = hipMalloc(&cudaOutput, arraySize * sizeof(int));
    if (c != hipSuccess) {
        printf("Failed to allocate cuda mem!\n");
        exit(1);
    }

    printf("starting!\n");

    testFunction <<<1, arraySize>>> (cudaOutput);    

    printf("ended!\n");

    c = hipGetLastError();
    if (c != hipSuccess) {
        printf("kernel error!\n");
        exit(1);
    }

    c = hipMemcpy(output, cudaOutput, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    if (c != hipSuccess) {
        printf("failed to copy!\n");
        printf("%s\n", hipGetErrorString(c));
        exit(1);
    }

    printf("\nOUTPUT\n");

    for (uint32_t i = 0; i < arraySize; i++) {
        printf("Thread %d: Output: %d\n", i, output[i]);
    }

    printf("at end\n");

    free(output);
    hipFree(cudaOutput);

}

__global__ void testFunction(int* outputArray) {
    int block = blockIdx.x + blockIdx.y * gridDim.x;
    int threadNumber = block * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;       
    
    //god help me
    rnd_state rnd_state;    
    bbRandom bb = bbRandom();
    bb.bbSeedRnd(&rnd_state, 100/*threadNumber*/);

    int a = threadNumber;        

    extern __shared__ RoomTemplates rts[roomTemplateAmount];

    //we want the first thread of each block to spawn the room templates;
    if (threadIdx.x == 0) {
        CreateRoomTemplates(rts);       
    }       

    __syncthreads();

    InitNewGame(&bb, &rnd_state, rts);

    outputArray[threadNumber] = a;      

    //TODO:
    //Once everything is working see if we can reduce the data type of some variables
    //from int32_t to int8_t or int16_t depending on the known max-limit of those variables.

    //See if we can just make a global rnd_state varaible instead of the stupid pointer stuff.

}
__device__ void dummy() {};